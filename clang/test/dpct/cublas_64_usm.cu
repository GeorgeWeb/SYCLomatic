// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// RUN: dpct --format-range=none --out-root %T/cublas_64_usm %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublas_64_usm/cublas_64_usm.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  int64_t ldc;
  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_s, *handle), A_s, lda, B_s, ldb, dpct::get_value(beta_s, *handle), C_s, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_d, *handle), A_d, lda, B_d, ldb, dpct::get_value(beta_d, *handle), C_d, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_c, *handle), (std::complex<float>*)A_c, lda, (std::complex<float>*)B_c, ldb, dpct::get_value(beta_c, *handle), (std::complex<float>*)C_c, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_z, *handle), (std::complex<double>*)A_z, lda, (std::complex<double>*)B_z, ldb, dpct::get_value(beta_z, *handle), (std::complex<double>*)C_z, ldc));
  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
}
